#include<stdio.h>
#include<hip/hip_runtime.h>


extern "C" 
{
    void mergeCuda(int* array, int startindex, int endindex, int threadCount);
}



__global__ void mergeKernel(int* subarray, int len, int sizetosort){
    unsigned int index = blockIdx.x *blockDim.x + threadIdx.x;
    int left = index*sizetosort;
    int right = left+sizetosort-1;
    while (left < len){
        // sizetosort is how many indices each thread is responsible for
        // size of left/right
        int lsize = sizetosort/2;
        int rsize = lsize;

        
        // overspill correction
        if (right >= len){
            right = len-1;
            int sublen = right-left+1;
            rsize = sublen-lsize;
            if (rsize < 0){
                rsize = 0;
            }
        }

        if (rsize != 0){
            printf("Sorting [%d,%d]\n",left,right);
            // sort
            int l = 0;
            int r = 0;
            while(l < lsize && r < rsize ){
                int lindex = l+left;
                int rindex = r+right;
                int leftV = subarray[lindex];
                int rightV = subarray[rindex];
                if (rightV < leftV){
                    subarray[lindex] = rightV;
                    subarray[rindex] = leftV;
                    r++;
                }
                else if(leftV < rightV){
                    l++;
                }
            }
        }
        index+=blockDim.x;
        left = index*sizetosort;
        right = left+sizetosort-1;
    }
    
}


void mergeCuda(int* array, int startindex, int endindex, int threadCount){
    // create the subarray
    int len = (endindex - startindex) + 1;
    int * subarray;
    hipMallocManaged(&subarray, len*sizeof(int));

    
    for (int i = 0; i < len; i++){
        subarray[i] = array[startindex+i];
    }

    printf("Original array: [");
    for (int i = 0; i < len-1; i++){
        printf("%d ",subarray[i]);
    }
    printf("%d]\n",subarray[len-1]);

    // kernel launch with this subarray    
    // sort every 2
    // sort every 4
    // sort every 8
    // ...

    int iteration = 1;
    int sizetosort = 0;
    while (sizetosort < len){
        sizetosort = 1 << iteration;
        int adjustedthreads = len/sizetosort;
        if (len%sizetosort > 0){
            adjustedthreads++;
        }
        if (adjustedthreads > threadCount){
            adjustedthreads = threadCount;
        }
        printf("Using %d threads\n", adjustedthreads);
        mergeKernel<<<1,adjustedthreads>>>(subarray, len, sizetosort);
        // finish this subsize, then loop
        hipDeviceSynchronize();
        printf("Resulting array: [");
        for (int i = 0; i < len-1; i++){
            printf("%d ",subarray[i]);
        }
        printf("%d]\n",subarray[len-1]);
        iteration++;
    }
    hipFree(subarray);
}