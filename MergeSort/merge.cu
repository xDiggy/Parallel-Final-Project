#include<stdio.h>
#include<hip/hip_runtime.h>


extern "C" 
{
    void mergeCuda(int* array, int startindex, int endindex, int threadCount);
}



__global__ void mergeKernel(int* subarray,int* subarray2, int len, int sizetosort){
    unsigned int index = blockIdx.x *blockDim.x + threadIdx.x;
    int left = index*sizetosort;
    int right = left+sizetosort-1;
    while (left < len){
        // sizetosort is how many indices each thread is responsible for
        // size of left/right
        int lsize = sizetosort/2;
        int rsize = lsize;

        
        // overspill correction
        if (right >= len){
            right = len-1;
            int sublen = right-left+1;
            rsize = sublen-lsize;
            if (rsize < 0){
                rsize = 0;
            }
        }

        if (rsize != 0){
            int added = 0;
            //printf("Sorting [%d,%d]\n",left,right);
            // sort
            int l = 0;
            int r = 0;
            while(l < lsize && r < rsize ){
                int lindex = l+left;
                int rindex = left+lsize+r;
                int leftV = subarray[lindex];
                int rightV = subarray[rindex];
                if (rightV == leftV){
                    subarray2[left+l+r]=rightV;
                    r++;
                    added++;
                    subarray2[left+l+r]=leftV;
                    l++;
                    added++;
                }
                if (rightV < leftV){
                    subarray2[left+l+r]=rightV;
                    //printf("1 placing %d at %d\n",rightV, left+l+r);
                    r++;
                    added++;
                }
                else if(leftV < rightV){
                    subarray2[left+l+r] = leftV;
                    //printf("2 placing %d at %d\n",leftV, left+l+r);
                    added++;
                    l++;
                }
            }
            if (l < lsize && r >= rsize){
                // add the rest of L to subarray2
                while (l < lsize){
                    int idex = left+added;
                    int lindex = l+left;
                    int leftV = subarray[lindex];
                    subarray2[idex] = leftV;
                    //printf("3 placing %d at %d\n",leftV, idex);
                    l++;
                    added++;
                } 
            }
            if(r < rsize && l >= lsize){
                // add the rest of R to subarray2
                while (r < rsize){
                    int idex = left+added;
                    int rindex = left+lsize+r;
                    int rightV = subarray[rindex];
                    subarray2[idex] = rightV;
                   // printf("4 placing %d at %d\n",rightV, idex);
                    r++;
                    added++;
                } 
            }
            // case for when things are equal!
        }
        else if (lsize > 0 && rsize == 0){
           // printf("Sorting [%d,%d]\n",left,right);
            for (int i = 0; i < lsize; i++){
                //printf("5 placing %d at %d\n",subarray[left+i], left+i);
                subarray2[left+i] = subarray[left+i];
            }
        }
        // there shouldn't be a case where left is 0 and right > 0 since we fill the left side first 
        index+=blockDim.x;
        left = index*sizetosort;
        right = left+sizetosort-1;
    }
    
}


void mergeCuda(int* array, int startindex, int endindex, int threadCount){
    // create the subarray
    int len = (endindex - startindex) + 1;
    int * subarray;
    int * subarray2;
    hipMallocManaged(&subarray, len*sizeof(int));
    hipMallocManaged(&subarray2, len*sizeof(int));

    
    for (int i = 0; i < len; i++){
        subarray[i] = array[startindex+i];
        subarray2[i] = array[startindex+i];
    }
    printf("Original array: [");
    for (int i = 0; i < len-1; i++){
        printf("%d ",subarray[i]);
    }
    printf("%d]\n",subarray[len-1]);

    // kernel launch with this subarray    
    // sort every 2
    // sort every 4
    // sort every 8
    // ...

    int iteration = 1;
    int sizetosort = 0;
    while (sizetosort < len){
        sizetosort = 1 << iteration;
        int adjustedthreads = len/sizetosort;
        if (len%sizetosort > 0){
            adjustedthreads++;
        }
        if (adjustedthreads > threadCount){
            adjustedthreads = threadCount;
        }
        mergeKernel<<<1,adjustedthreads>>>(subarray,subarray2, len, sizetosort);
        hipDeviceSynchronize();

        // finish this subsize, copy, then loop
        hipMemcpy(subarray, subarray2, len * sizeof(int), hipMemcpyDeviceToDevice);

        for (int i = 0; i < len; i++){
            subarray2[i] = -1;
        }
        iteration++;        
    }
    printf("Resulting array: [");
    for (int i = 0; i < len-1; i++){
        printf("%d ",subarray[i]);
    }
    printf("%d]\n",subarray[len-1]);
    
    hipFree(subarray);
    hipFree(subarray2);
}