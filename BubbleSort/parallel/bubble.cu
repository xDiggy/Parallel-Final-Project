#include<stdio.h>
#include<hip/hip_runtime.h>


extern "C"
{
    void bubbleCuda(int* array, int len);
}

__global__ void bubbleKernel(int* array, int len){

    int threadID = threadIdx.x;

    for(int i = 0; i < len; i++){

        int offset = i % 2;
        int index = threadID * 2 + offset;

        if(index < len - 1){
            if(array[index] > array[index + 1]){
                int temp = array[index];
                array[index] = array[index + 1];
                array[index + 1] = temp;
            }
        }
        __syncthreads();
    }

}

void bubbleCuda(int* array, int len){

    int *d_array;
    hipMalloc((void**)&d_array, len * sizeof(int));
    hipMemcpy(d_array, array, len * sizeof(int), hipMemcpyHostToDevice);

    bubbleKernel<<<1, len/2>>>(d_array, len);
    hipDeviceSynchronize();

    hipMemcpy(array, d_array, len * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_array);

}